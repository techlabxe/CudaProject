#include <stdio.h>
#include <hip/hip_runtime.h>

#include <chrono>

using namespace std;

__global__
void addKernel( int* a, int* b, int* c )
{
    int idx = threadIdx.x;
    c[idx] = a[idx] + b[idx];
}
void testCudaAdd() {
    const int COUNT = 10;
    int* a = new int[COUNT];
    int* b = new int[COUNT];
    int* c = new int[COUNT];

    for( int i=0;i<COUNT;++i) {
        a[i] = i;
        b[i] = i*100;
    }

    int *devArrayA = 0, *devArrayB = 0, *devArrayC = 0;
    hipMalloc( &devArrayA, sizeof(int) * COUNT );
    hipMalloc( &devArrayB, sizeof(int) * COUNT );
    hipMalloc( &devArrayC, sizeof(int) * COUNT );

    // ���̓f�[�^�̓]��.
    hipMemcpy( devArrayA, a, sizeof(int)*COUNT, hipMemcpyHostToDevice );
    hipMemcpy( devArrayB, b, sizeof(int)*COUNT, hipMemcpyHostToDevice );

    // ���s.
    addKernel<<<1, COUNT>>>(devArrayA, devArrayB, devArrayC );
    hipDeviceSynchronize();

    // ���ʂ̓ǂݖ߂�.
    hipMemcpy( c, devArrayC, sizeof(int)*COUNT, hipMemcpyDeviceToHost );
    for(int i=0;i<COUNT;++i) {
        printf( "%d ", c[i] );
    }
    printf( "\n" );

    hipFree( devArrayC );
    hipFree( devArrayB );
    hipFree( devArrayA );

    delete[] a;
    delete[] b;
    delete[] c;
}

__global__
void multMatrix( float* a, float* b, float* c, int COUNT )
{
    int idx = blockDim.x * threadIdx.y + threadIdx.x;
    int idxCol = blockDim.x * blockIdx.x + threadIdx.x;
    int idxRow = blockDim.y * blockIdx.y + threadIdx.y;
    float scanSum = 0;
    for( int i=0;i<COUNT;++i) {
        if( idxCol >= COUNT || idxRow >= COUNT ) {
            continue;
        }
#if 01
        scanSum += a[ idxRow*COUNT + i ] * b[ idxCol + i*COUNT ]; 
#else
        // ���x�̖�肪�o����.
        scanSum = __fadd_rn( scanSum, __fmul_rn( a[idxRow*COUNT+i], b[idxCol+i*COUNT] ) );
#endif
    }
    if( idxCol < COUNT && idxRow < COUNT ) {
        c[idxCol+idxRow*COUNT] = scanSum;
    }
}
void testCudaMult() {
    const int COUNT = 1024;
    const int SIZE = COUNT*COUNT; // �s��T�C�Y
    float* a = new float[SIZE];
    float* b = new float[SIZE];
    float* c = new float[SIZE];
    for( int i=0;i<SIZE;++i) {
        a[i] = float(0.001f * i );
        b[i] = float(0.005f * i );
    }

    chrono::high_resolution_clock::time_point start, stop;
    start = chrono::high_resolution_clock::now();
    for( int i=0;i<COUNT;++i ) {
        for( int j=0;j<COUNT;++j ) {
            float tmp = float(0);
            for(int t=0;t<COUNT;++t) {
                tmp += a[t+i*COUNT] * b[j+COUNT*t];
            }
            c[i*COUNT+j] = tmp;
        }
    }
    stop = chrono::high_resolution_clock::now();

#if 0
    for(int i=0;i<COUNT;++i) {
        for(int j=0;j<COUNT;++j) {
            printf( "%f ", c[i*COUNT+j] );
        }
        printf( "\n" );
    }
#endif
    chrono::microseconds cpuTime = chrono::duration_cast<chrono::microseconds>(stop-start);
    printf( "CPU: %d (us)\n", cpuTime.count() );
    printf( "\n" );

    float* gpuC = new float[SIZE];
    for( int i=0;i<SIZE;++i) {
        a[i] = float(0.001f * i );
        b[i] = float(0.005f * i );
        gpuC[i] = 0.0f;
    }

    float *devA, *devB, *devC;
    hipMalloc( &devA, sizeof(float) * SIZE );
    hipMalloc( &devB, sizeof(float) * SIZE );
    hipMalloc( &devC, sizeof(float) * SIZE );
    hipMemcpy( devA, a, sizeof(float)*SIZE, hipMemcpyHostToDevice );
    hipMemcpy( devB, b, sizeof(float)*SIZE, hipMemcpyHostToDevice );
    
    start = chrono::high_resolution_clock::now();

    const int thrCount=32;
    int blockXY = ( COUNT + (thrCount-1) ) / thrCount;
    dim3 blk( blockXY,blockXY);
    dim3 thr( thrCount,thrCount);
    multMatrix<<<blk,thr>>>( devA, devB, devC, COUNT );
    hipDeviceSynchronize();
    stop = chrono::high_resolution_clock::now();
    hipMemcpy( gpuC, devC, sizeof(float)*SIZE, hipMemcpyDeviceToHost );
#if 0
    for(int i=0;i<COUNT;++i) {
        for(int j=0;j<COUNT;++j) {
            printf( "%.2f(%.2f) ", gpuC[i*COUNT+j], c[i*COUNT+j] );
        }
        printf( "\n" );
    }
#endif
    int mismatchCount = 0;
    for(int i=0;i<SIZE;++i) {
        if( c[i] != gpuC[i] ) {
            mismatchCount++;
        }
    }
    if( mismatchCount > 0 ) {
        printf( "mismatchCount= %d (%d)\n", mismatchCount, SIZE );
    }

    chrono::microseconds gpuTime = chrono::duration_cast<chrono::microseconds>(stop-start);
    printf( "GPU: %d (us)\n", gpuTime.count() );
    printf( "\n" );
    printf( "rate = %.4f (%dx%d matrix)\n", cpuTime.count() / (double)gpuTime.count(), COUNT, COUNT );
}

int main() {
    printf( "Hello,CUDA\n" );
    testCudaAdd();
    testCudaMult();
    hipDeviceReset();
    return 0;
}
